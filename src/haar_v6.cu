#include "hip/hip_runtime.h"
/*
*  haar_vX is the well synchronized version of haar transform into de GPU.
*  This version work with all the samples as a matrix into de GPU
*  with dimension SAMPLES x (sample_num + data_adjust) (rows x cols)
*  Copyright (C) 2018 Lisardo Fernández Cordeiro <lisardo.fernandez@uv.es>
*
*  This program is free software; you can redistribute it and/or modify
*  it under the terms of the GNU General Public License as published by
*  the Free Software Foundation; either version 2, or (at your option)
*  any later version.
*
*  This program is distributed in the hope that it will be useful,
*  but WITHOUT ANY WARRANTY; without even the implied warranty of
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*  GNU General Public License for more details.
*
*  You should have received a copy of the GNU General Public License
*  along with this program; if not, write to the Free Software
*  Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA 02110-1301 USA.
*
*/

/** \file
*  \brief Archivo para procesamiento de diferentes muestras metiladas de ADN.
*
*  Este archivo contiene la definición de las funciones para:
*         ..carga de datos en GPU
*         ..lanzamiento de proceso de transformación en GPU
*         ..kernel en GPU para control de transformación en niveles definidos
*         ..kernel de transformación wavelet del vector seleccionado
*         ..kernel para copiar coeficientes desde vector auxiliar de sincronización
*/

#include <stdio.h>
#include <GL/gl.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "data_pack.h"

#define BLOCK_SIZE  1024		// número de hilos por bloque de GPU
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); } // para gestión de errores en GPU



/** ***********************************************************************************************
  * \fn void gpuAssert(hipError_t, char*, int, bool)
  *  \brief Función responsable de recoger error en GPU y mostrarlo
  *  \param code	código de error de la GPU
  *  \param *file	fichero donde se produce el error
  *  \param line	línea de código donde se produce el error
  *  \param abort	indica si se sale del programa
  * ***********************************************************************************************
  */
extern "C"
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

/** ***********************************************************************************************
  * \fn void copyValuesTotal(float*, float *, int, int)
  *  \brief función "hija" en GPU responsable de la copia de los datos del segmento a transformar
  *         proporcionando sincronización a nivel GRID
  *  \param *haar	puntero a vector de datos original
  *  \param *aux	puntero a vector de datos a transformar
  *  \param num		numero datos totales a transformar
  *  \param pi  	posicion inicial de copia - offset
  * ***********************************************************************************************
  */
extern "C"
__global__
void copyValuesTotal(float *haar, float *aux, int num, int posicion_inicial)
{
    // variables ------------------------------------------------------------------------------
    int index = threadIdx.x + blockIdx.x * blockDim.x;	// índice sobre todo el vector

    // copiar todos los valores de haar en aux
    if (index < num)
            aux[index] = haar[index + posicion_inicial];
}


/** ***********************************************************************************************
  * \fn void copyValues(float*, float *, int)
  *  \brief función "hija" en GPU responsable de la copia de los valores escalados
  *         proporcionando sincronización a nivel GRID
  *  \param *aux	puntero a vector de datos a transformar
  *  \param *temp	puntero a vector de datos temporales a copiar
  *  \param num     numero datos a copiar
  * ***********************************************************************************************
  */
extern "C"
__global__
void copyValues(float *aux, float *temp, int num)
{
    // variables ----------------------------------------------------------------------------------
    int index = threadIdx.x + blockIdx.x * blockDim.x;	// índice sobre todo el vector

    // copiar todos los valores de haar en aux
    if (index < num)
        aux[index] = temp[index];
}


/** ***********************************************************************************************
  * \fn void transform(float*, int, int, int)
  *  \brief función "hija" en GPU responsable de la transformación wavelet de un vector
  *         proporcionando sincronización a nivel GRID.
  *  \param *aux	puntero a vector de datos a transformar
  *  \param *temp	puntero a vector de resultados intermedios
  *  \param num		número de posiciones del vector
  * ***********************************************************************************************
  */
extern "C"
__global__
void transform(float *aux, float *temp, int num)
{
    // variables ----------------------------------------------------------------------------------
    int index = threadIdx.x + blockIdx.x * blockDim.x;	// índice sobre todo el vector
    float f   = 0.7071067811865476;                     // coeficiente haar wavelet
    float aux1;                                         // variables auxiliares de sincronización
    int idx;                                            // indice auxiliar para guardar dato

    // transformada haar en paralelo sobre el vector recibido -------------------------------------
    if (index < num)
    {
        if ((index & 0x01) == 0)	// solo los hilos con índice par (0, 2, 4, ...)
        {
            idx = index * 0.5;

            aux1 = (aux[index] + aux[index + 1]) * f;	// escalado (filtro paso-bajo)

            temp[idx]  = aux1;
        }
    }
}


/** ***********************************************************************************************
  * \fn void wavedec(float*, float**, int, int, int, int, int)
  *  \brief Función principal en GPU responsable de calcular y ordenar las partes del vector
  *         para su transformación wavelet multinivel.
  *  \param *haar	puntero a matriz de datos a transformar
  *  \param *aux	puntero a matriz de coeficiente auxiliares para ayuda a la sincronización
  *  \param pitch	desplazamiento óptimo en memoria GPU para alojar cada muestra
  *  \param pitch_2	desplazamiento óptimo en memoria GPU para alojar cálculo auxiliar
  *  \param n		número total de posiciones del vector
  *  \param l		número de niveles a computar
  *  \param samples número de muestras a analizar
  *  \param pi      posición inicial del segmento de datos a analizar
  * ***********************************************************************************************
  */
extern "C"
__global__
void wavedec(float *haar, float *aux, float *temp,
             size_t pitch, size_t pitch_2, size_t pitch_3,
             int n, int l, int samples, int pi)
{
    // variables ----------------------------------------------------------------------------------
    int index_X = threadIdx.x + blockIdx.x * blockDim.x;	// indice de hilos sobre todo el vector
    int level   = 0;                                        // número de nivel
    int num     = n;                                        // número de posiciones en vector
    int hilo;                                               // guarda el hilo asignado para que se resposabilice de todo el proceso

    // limita el número de hilos al de muestras ---------------------------------------------------
    if (index_X < samples)
    {
        hilo = index_X;		// cada hilo se responsabiliza de una misma muestra

        if (hilo == index_X)
        {
            // separar los datos por muestras - - - - - - - - - - - - - - - - - - - - - - - - - - -
            float *haar_c = (float *)((char *)haar + index_X * pitch);
            float *aux_c  = (float *)((char *)aux  + index_X * pitch_2);
            float *temp_c = (float *)((char *)temp + index_X * pitch_3);

            __syncthreads();

            // llamada a función hija para copiar segmento de vector a transformar
            copyValuesTotal<<<(num + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(haar_c,
                                                                               aux_c,
                                                                               num,
                                                                               pi);


            // procesamiento multinivel del vector de datos ---------------------------------------
            // repite la transformación tantas veces como niveles se han solicitado
            while (level < l && num >= 2)
            {
                // llamada a función hija para transformación del nivel correspondiente
                // con esta división en padre-hijo, se consigue sincronizar cada nivel
                // \param	<<<((datos_x_muestra + num_hilos_bloque-1) / num_hilos_bloque),
                // 		numero hilos por bloque>>>
                transform<<<(num + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(aux_c,
                                                                             temp_c,
                                                                             num);


                // actualizar variables de nivel  - - - - - - - - - - - - - - - - - - - - - - - - -
                level += 1;
                num    = ceilf(num * 0.5);


                // llamada a función hija para copiar resultados en vector auxiliar
                copyValues<<<(num + BLOCK_SIZE-1) / BLOCK_SIZE, BLOCK_SIZE>>>(aux_c,
                                                                              temp_c,
                                                                              num);


                // actualiza el número de datos para el siguiente nivel - - - - - - - - - - - - - -
                if ((num & 01) == 1)
                {
                    num++;
                    aux_c[num] = 0;
                }
            }
        }
    }
}

/** ***********************************************************************************************
  * \fn void cuda_send_data(datos_cuda &)
  *  \brief Función para enviar los datos a la GPU
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_send_data(datos_cuda &cuda_data)
{
    // reserva espacio en GPU para el vector a transformar y copia matriz de datos ----------------
    // devuelve valor de desplazamiento (pitch) óptimo para gestión de memoria adecuada
    // en función de la cantdad de datos a alojar
    // \param 	puntero a posición memoria GPU,
    //          desplazamiento óptimo devuelto por CUDA,
    //          cantidad de bytes a reservar por fila,
    //          número de muestras (filas)
    gpuErrchk(hipMallocPitch(&cuda_data.d_haar,
                              &cuda_data.pitch,
                              (cuda_data.sample_num + cuda_data.data_adjust) * sizeof(float),
                              cuda_data.samples));


    gpuErrchk(hipMallocPitch(&cuda_data.d_aux,
                              &cuda_data.pitch_2,
                              (cuda_data.sample_num + cuda_data.data_adjust) * sizeof(float),
                              cuda_data.samples));


    // envío de datos a GPU -----------------------------------------------------------------------
    // \param	puntero a posición de memoria GPU,
    //          desplazamiento óptimo,
    //          puntero a posición de datos en CPU a enviar a GPU,
    //          cantidad de bytes a enviar por muestra,
    //          cantidad de bytes a alojar por muestra,
    //          número de filas (muestras)
    gpuErrchk(hipMemcpy2D( cuda_data.d_haar,
                            cuda_data.pitch,
                            cuda_data.mc_full[0],
                            cuda_data.sample_num * sizeof(float),
                            cuda_data.sample_num * sizeof(float),
                            cuda_data.samples,
                            hipMemcpyHostToDevice));

}


/** ***********************************************************************************************
  * \fn void cuda_main(datos_cuda &)
  *  \brief Función para procesar los datos en la GPU
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_main(datos_cuda &cuda_data)
{
    // reserva TODA la memoria CONTIGUA para la matriz de muestras tranformadas -------------------
    // para trasvase de datos entre GPU y CPU con CUDA, la matriz debe ser contigua completa
    cuda_data.h_haar_C = new float*[cuda_data.samples];                             // reservar punteros a filas
    cuda_data.h_haar_C[0] = new float[cuda_data.samples * (cuda_data.h_haar_L[0])];	// reservar toodos los datos (rows * cols)
    for (int i = 1; i < cuda_data.samples; i++)                                     // asignar valor a punteros de fila
        cuda_data.h_haar_C[i] = cuda_data.h_haar_C[i - 1] + cuda_data.h_haar_L[0];


    // reserva memoria para cálculos temporales en GPU --------------------------------------------
    float *d_temp;
    size_t pitch;
    gpuErrchk(hipMallocPitch(&d_temp,
                              &pitch,
                              (cuda_data.sample_num + 1) * 0.7 * sizeof(float),
                              cuda_data.samples));


    // transforma el número de muestras elegida ---------------------------------------------------
    // realiza la transformación en la GPU del conjunto de muestras cargado
    // \param	<<< número de bloques a utilizar,
    //          número de hilos por bloque >>> (máximo 1024 para PASCAL GTX 1080)
    // \param	puntero a datos a transformar alojados en GPU,
    //          desplazamiento óptimo de datos por fila,
    //          número de datos por muestra (fila) a transformar,
    //          ajuste de longitud de muestra por número impar al dividir la muestra
    wavedec<<<1, cuda_data.samples>>>(cuda_data.d_haar,
                                      cuda_data.d_aux,
                                      d_temp,
                                      cuda_data.pitch,
                                      cuda_data.pitch_2,
                                      pitch,
                                      cuda_data.sample_num,
                                      cuda_data.levels,
                                      cuda_data.samples,
                                      cuda_data.rango_inferior);

    // espera a que la GPU termine el trabajo - - - - - - - - - - - - - - - - - - - - - - - - - - -
    gpuErrchk(hipDeviceSynchronize());


    // recupera el resultado de la transformación en memoria GPU a memoria CPU- - - - - - - - - - -
    // \param	puntero a matriz de datos a guardar en CPU,
    //          cantidad de bytes a guardar por muestra,
    //          puntero a datos para copiar de GPU,
    //          desplazamiento óptimo de datos por fila en GPU,
    //          cantidad de bytes en GPU a copiar por muestra,
    //          número de muestras (filas)
    gpuErrchk(hipMemcpy2D(	cuda_data.h_haar_C[0],
                            cuda_data.h_haar_L[0] * sizeof(float),
                            cuda_data.d_aux,
                            cuda_data.pitch,
                            cuda_data.h_haar_L[0] * sizeof(float),
                            cuda_data.samples,
                            hipMemcpyDeviceToHost));


    //libera la memoria temporal utilizada para cálculos intemedios
    hipFree(d_temp);
}

/** ***********************************************************************************************
  * \fn void *cuda_init()
  *  \brief Función para inicializar la gpu
  * ***********************************************************************************************
  */
void cuda_init()
{
    int deviceCount = 0;
    int cudaDevice  = 0;
    char cudaDeviceName [100];
    hipDeviceProp_t prop;
    hipInit(0);
    hipGetDeviceCount(&deviceCount);
    hipDeviceGet(&cudaDevice, 0);
    hipDeviceGetName(cudaDeviceName, 100, cudaDevice);
    hipGetDeviceProperties(&prop, cudaDevice);

    if (hipChooseDevice(&cudaDevice, &prop) != hipSuccess)
        puts("failed to choose device");
    if (cudaGLSetGLDevice(cudaDevice) != hipSuccess)
        puts("failed to set gl device");

    printf("Number of devices: %u \t cuda device: %d\n", deviceCount, cudaDevice);
    printf("Device name: %s\n", cudaDeviceName);
    printf("Warp size: %u\n", prop.warpSize);
}

/** ***********************************************************************************************
  * \fn void cuda_end(data buf)
  *  \brief Función para liberar memoria de la GPU
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_end(datos_cuda &cuda_data)
{
    //libera la memoria de la gpu utilizada para cálculos intemedios
    hipFree(cuda_data.d_haar);
    hipFree(cuda_data.d_aux);
}

/** ***********************************************************************************************
  * \fn void calculo_haar_L(datos_cuda &cuda_data)
  *  \brief Función para calcular el número de datos en el nivel dado y el ajuste por impares
  *  \param &cuda_data  estructura con variables de control de datos
  * ***********************************************************************************************
  */
void cuda_calculo_haar_L(datos_cuda &cuda_data)
{
    // cálculo de número de coeficientes por nivel y del ajuste de paso entre escala y coeficiente
    cuda_data.h_haar_L.push_front(cuda_data.sample_num);	// última posición guarda el total de posiciones por muestra

    // para cada nivel se divide por dos la cantidad de posiciones del nivel anterior -------------
    // redondeando al alza y actualizando el ajuste cuando sea impar
    for (int fila = cuda_data.levels; fila > 0; fila--)
    {
        if (ceil(cuda_data.h_haar_L.front() * 0.5 >= 2))
        {
            cuda_data.h_haar_L.push_front(ceil(cuda_data.h_haar_L.front() * 0.5));
            if (fila > 0 && cuda_data.h_haar_L[1] != cuda_data.sample_num)
                cuda_data.data_adjust += size_t(2 * cuda_data.h_haar_L.front() - cuda_data.h_haar_L[1]);
        }
        else
            break;
    }
    cuda_data.h_haar_L.push_front(cuda_data.h_haar_L.front());	// primera posición coincide con el número de datos de escala
}
